#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define SIZE 8

__global__ void max(int *input)
{
    const int tid=threadIdx.x;
    int stepsize=1;
    int numThreads=blockDim.x;
    
    while(numThreads>0)
    {
        if(tid<numThreads)
        {
            int first=tid*stepsize*2;
            int second=first+stepsize;
            input[first]+=input[second];
        }
        
        stepsize <<=1;
        numThreads >>=1;
    }
    
}

int main()
{
    srand(time(NULL));
    int a[SIZE],result;
    int *dev_a;
    
    for(int i=0;i<SIZE;i++)
    {
        a[i]=rand()%50;
    }
    
    printf(" Input : ");
    for(int i=0;i<SIZE;i++)
    {
        printf("%d ",a[i]);
    }
    
    hipMalloc(&dev_a,SIZE*sizeof(int));
    
    hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
    max<<<1,SIZE/2>>>(dev_a);
    hipMemcpy(&result,dev_a,sizeof(int),hipMemcpyDeviceToHost);
    
    printf(" Output %d",result);
    
}