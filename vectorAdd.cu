#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define SIZE 8

__global__ void vectAdd(int *a,int *b,int *c)
{
    const int tid=threadIdx.x;
    c[tid]=a[tid]+b[tid];
    
}

int main()
{
    srand(time(NULL));
    int a[SIZE],b[SIZE],c[SIZE];
    int *dev_a,*dev_b,*dev_c;
    
    for(int i=0;i<SIZE;i++)
    {
        a[i]=rand()%50;
        b[i]=rand()%50;
        c[i]=0;
    }
    
    printf(" Input A : ");
    for(int i=0;i<SIZE;i++)
    {
        printf("%d ",a[i]);
    }
    
    printf(" Input B : ");
    for(int i=0;i<SIZE;i++)
    {
        printf("%d ",b[i]);
    }
    
    hipMalloc(&dev_a,SIZE*sizeof(int));
    hipMalloc(&dev_b,SIZE*sizeof(int));
    hipMalloc(&dev_c,SIZE*sizeof(int));
    
    hipMemcpy(dev_a,a,SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,SIZE*sizeof(int),hipMemcpyHostToDevice);
    vectAdd<<<1,SIZE>>>(dev_a,dev_b,dev_c);
    hipMemcpy(&c,dev_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
    
    printf(" Output : ");
    for(int i=0;i<SIZE;i++)
    {
        printf("%d ",c[i]);
    }
    
}