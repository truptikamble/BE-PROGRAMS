#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define SIZE 3

__global__ void multi(int *vect,int *mat,int *result)
{
    int tid=blockDim.x*blockIdx.x;
    int value=0;
    for(int i=0;i<SIZE;i++)
    {
        value=value+(mat[i+tid]*vect[i]);
    }
    result[blockIdx.x]=value;
}
int main()
{
    srand(time(NULL));
    int a[SIZE][SIZE],b[SIZE],c[SIZE];
    int *dev_a,*dev_b,*dev_c;
    
    for(int i=0;i<SIZE;i++)
    {
        for(int j=0;j<SIZE;j++)
        {
          a[i][j]=rand()%50;
        }
    }
    
    for(int i=0;i<SIZE;i++)
    {
        b[i]=rand()%50;
        
    }
    
     printf("input A:");
    for(int i=0;i<SIZE;i++)
    {
       for(int j=0;j<SIZE;j++)
        {
          printf("%d ",a[i][j]);
        }
    }
    
    
    printf(" Input B : ");
    for(int i=0;i<SIZE;i++)
    {
        printf("%d ",b[i]);
    }
    
    hipMalloc(&dev_a,SIZE*SIZE*sizeof(int));
    hipMalloc(&dev_b,SIZE*sizeof(int));
    hipMalloc(&dev_c,SIZE*sizeof(int));
    
    hipMemcpy(dev_a,a,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,SIZE*sizeof(int),hipMemcpyHostToDevice);
    multi<<<SIZE,SIZE>>>(dev_b,dev_a,dev_c);
    hipMemcpy(&c,dev_c,SIZE*sizeof(int),hipMemcpyDeviceToHost);
    
    printf(" Output : ");
    for(int i=0;i<SIZE;i++)
    {
        printf("%d ",c[i]);
    }
    
}