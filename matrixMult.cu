#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define SIZE 3

__global__ void multi(int *Md, int *Nd, int *Pd) {
    //2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    int Pvalue = 0;

    for(int k = 0; k <SIZE ; ++k)
    {
        int Mdelement = Md[ty*SIZE + k];
        int Ndelement = Nd[k*SIZE + tx];
        Pvalue += (Mdelement*Ndelement);
    }

    Pd[ty*SIZE + tx] = Pvalue;
}
int main()
{
    srand(time(NULL));
    int a[SIZE*SIZE],b[SIZE*SIZE],c[SIZE*SIZE];
    int *dev_a,*dev_b,*dev_c;
    
    for(int i=0;i<SIZE*SIZE;i++)
    {
        a[i]=rand()%50;
        b[i]=rand()%50;
     
    }
    
    
     printf("input A:");
    for(int i=0;i<SIZE*SIZE;i++)
    {
       
          printf("%d ",a[i]);
        
    }
    
    printf("input B:");
    for(int i=0;i<SIZE*SIZE;i++)
    {
       
          printf("%d ",b[i]);
        
    }
    
    dim3 dimGrid(1,1);
    dim3 dimBlock(3,3);
    
    hipMalloc(&dev_a,SIZE*SIZE*sizeof(int));
    hipMalloc(&dev_b,SIZE*SIZE*sizeof(int));
    hipMalloc(&dev_c,SIZE*SIZE*sizeof(int));
    
    hipMemcpy(dev_a,a,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_b,b,SIZE*SIZE*sizeof(int),hipMemcpyHostToDevice);
    multi<<<dimGrid,dimBlock>>>(dev_a,dev_b,dev_c);
    hipMemcpy(&c,dev_c,SIZE*SIZE*sizeof(int),hipMemcpyDeviceToHost);
    
    printf("Output :");
    for(int i=0;i<SIZE*SIZE;i++)
    {
       
          printf("%d ",c[i]);
        
    }
    
}